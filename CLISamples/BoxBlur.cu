#include <hip/hip_runtime.h>

__global__ void boxBlurKernel(uchar4* pDataIn, uchar4* pDataOut, 
    int width, int height, int borderSize)
{
    // get the position for the current thread
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    // calculate the memory adress
    const int tid = y * width + x;

    uchar4 value = {127, 127, 127, 255};

    if(x >= borderSize && y >= borderSize 
    && x + borderSize < width && y + borderSize < height)
    {
        int denom = 2 * borderSize + 1;
        denom *= denom;
    
        int3 sum = {0, 0, 0};
    
        for(int dy = -borderSize; dy <= borderSize; dy++)
        for(int dx = -borderSize; dx <= borderSize; dx++)
        {
            uchar4 locValue = pDataIn[(y + dy) * width + (x + dx)];

            sum.x += locValue.x;
            sum.y += locValue.y;
            sum.z += locValue.z;
        }

        value.x = (unsigned char)(sum.x / denom);
        value.y = (unsigned char)(sum.y / denom);
        value.z = (unsigned char)(sum.z / denom);
    }

    if(x < width && y < height)
    {
        // write the value back to the global memory
        pDataOut[tid] = value;
    }
}

void boxBlur(uchar4* pDataIn, uchar4* pDataOut, 
    int width, int height, 
    int blurSize, int blockDimX, int blockDimY)
{
    // allocate device memory
    unsigned int mem_size = sizeof(uchar4) * width * height;

    uchar4* pDevDataIn;
    uchar4* pDevDataOut;

    hipError_t res;
    res = hipMalloc((void **) &pDevDataIn, mem_size);
    res = hipMalloc((void **) &pDevDataOut, mem_size);

    // copy results from host to device
    res = hipMemcpy(pDevDataIn, pDataIn, mem_size, hipMemcpyHostToDevice);

    // define partitioning
    dim3 threadsPerBlock(blockDimX, blockDimY);
    dim3 numBlocks(width / threadsPerBlock.x + 1, height / threadsPerBlock.y + 1);

    // run the cuda kernel 
    boxBlurKernel<<<numBlocks, threadsPerBlock>>>(pDevDataIn, pDevDataOut, 
        width, height, blurSize);

    // copy results from device to host
    res = hipMemcpy(pDataOut, pDevDataOut, mem_size, hipMemcpyDeviceToHost);

    // cleanup memory
    res = hipFree(pDevDataIn);
    res = hipFree(pDevDataOut);
}
