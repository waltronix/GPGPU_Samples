#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

texture<uchar4, 2, hipReadModeElementType> texIn;

__global__ void boxBlurTexture2DKernel(uchar4* pData, int width, int height, int borderSize)
{
    // get the position for the current thread
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    // calculate the memory adress
    const int tid = y * width + x;

    uchar4 value = {127, 127, 127, 255};

    int denom = 2 * borderSize + 1;
    denom *= denom;
    
    int3 sum = {0, 0, 0};
    
    for(int dy = -1 * borderSize; dy <= borderSize; dy++)
    for(int dx = -1 * borderSize; dx <= borderSize; dx++)
    {
        uchar4 locValue = tex2D(texIn, x + dx, y + dy);

        sum.x += locValue.x;
        sum.y += locValue.y;
        sum.z += locValue.z;
    }

    value.x = (unsigned char)(sum.x / denom);
    value.y = (unsigned char)(sum.y / denom);
    value.z = (unsigned char)(sum.z / denom);
    
    if(x < width && y < height)
    {
        // write the value back to the global memory
        pData[tid] = value;
    }
}

void boxBlurTexture2D(uchar4* pDataIn, uchar4* pDataOut, int width, int height, int blurSize)
{
    // allocate device memory
    uchar4* pDevDataIn;
    uchar4* pDevDataOut;
    unsigned int mem_size = sizeof(uchar4) * width * height;

    hipError_t res;
    res = hipMalloc((void **) &pDevDataIn, mem_size);
    res = hipMemcpy(pDevDataIn, pDataIn, mem_size, hipMemcpyHostToDevice);

    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    res = hipBindTexture2D(NULL, texIn, pDevDataIn, desc, width, height, sizeof(uchar4) * width);
    
    res = hipMalloc((void **) &pDevDataOut, mem_size);

    // define partitioning
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(
        width / threadsPerBlock.x + 1, 
        height / threadsPerBlock.y + 1);

    // run the cuda kernel 
    boxBlurTexture2DKernel<<<numBlocks, threadsPerBlock>>>(pDevDataOut, width, height, blurSize);

    // copy results from device to host
    res = hipMemcpy(pDataOut, pDevDataOut, mem_size, hipMemcpyDeviceToHost);

    // cleanup memory
    res = hipUnbindTexture(texIn);
    res = hipFree(pDevDataIn);
    res = hipFree(pDevDataOut);
}
