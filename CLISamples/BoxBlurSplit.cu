#include <hip/hip_runtime.h>

__global__ void boxBlurSplitXKernel(uchar4* pDataIn, uchar4* pDataOut, 
    int width, int height, int borderSize)
{
    // get the position for the current thread
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    // calculate the memory adress
    const int tid = y * width + x;

    uchar4 value = {127, 127, 127, 255};

    if(x >= borderSize && x + borderSize < width)
    {
        int denom = 2 * borderSize + 1;
    
        int3 sum = {0, 0, 0};
    
        for(int dx = -borderSize; dx <= borderSize; dx++)
        {
            uchar4 locValue = pDataIn[y * width + (x + dx)];

            sum.x += locValue.x;
            sum.y += locValue.y;
            sum.z += locValue.z;
        }

        value.x = (unsigned char)(sum.x / denom);
        value.y = (unsigned char)(sum.y / denom);
        value.z = (unsigned char)(sum.z / denom);
    }

    if(x < width && y < height)
    {
        // write the value back to the global memory
        pDataOut[tid] = value;
    }
}

__global__ void boxBlurSplitYKernel(uchar4* pDataIn, uchar4* pDataOut, 
    int width, int height, int borderSize)
{
    // get the position for the current thread
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    // calculate the memory adress
    const int tid = y * width + x;

    uchar4 value = {127, 127, 127, 255};

    if(y >= borderSize && y + borderSize < height)
    {
        int denom = 2 * borderSize + 1;
    
        int3 sum = {0, 0, 0};
    
        for(int dy = -borderSize; dy <= borderSize; dy++)
        {
            uchar4 locValue = pDataIn[(y + dy) * width + x];

            sum.x += locValue.x;
            sum.y += locValue.y;
            sum.z += locValue.z;
        }

        value.x = (unsigned char)(sum.x / denom);
        value.y = (unsigned char)(sum.y / denom);
        value.z = (unsigned char)(sum.z / denom);
    }

    if(x < width && y < height)
    {
        // write the value back to the global memory
        pDataOut[tid] = value;
    }
}

void boxBlurSplit(uchar4* pDataIn, uchar4* pDataOut, 
    int width, int height, int blurSize, int blockDimX, int blockDimY)
{
    // allocate device memory
    unsigned int mem_size = sizeof(uchar4) * width * height;

    uchar4* pDevDataIn;
    uchar4* pDevDataOut;

    hipError_t res;
    res = hipMalloc((void **) &pDevDataIn, mem_size);
    res = hipMalloc((void **) &pDevDataOut, mem_size);

    // copy results from host to device
    res = hipMemcpy(pDevDataIn, pDataIn, mem_size, hipMemcpyHostToDevice);

    // define partitioning
    dim3 threadsPerBlock(blockDimX, blockDimY);
    dim3 numBlocks(width / threadsPerBlock.x + 1, height / threadsPerBlock.y + 1);

    // run the cuda kernel 
    boxBlurSplitXKernel<<<numBlocks, threadsPerBlock>>>(
        pDevDataIn, pDevDataOut, width, height, blurSize);
    
    boxBlurSplitYKernel<<<numBlocks, threadsPerBlock>>>(
        pDevDataOut, pDevDataIn, width, height, blurSize);

    // copy results from device to host
    res = hipMemcpy(pDataOut, pDevDataIn, mem_size, hipMemcpyDeviceToHost);

    // cleanup memory
    res = hipFree(pDevDataIn);
    res = hipFree(pDevDataOut);
}
